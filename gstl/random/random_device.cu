#include "hip/hip_runtime.h"
#include <gstl/random/random_device.cuh>

#include <gstl/utility/hash.cuh>
#include <gstl/utility/limits.cuh>

namespace gpu
{
	inline GPU_DEVICE double random_device::entropy() const noexcept
	{
		return 0.0;
	}

	inline GPU_DEVICE GPU_CONSTEXPR random_device::result_type random_device::max()
	{
		return numeric_limits<result_type>::max();
	}

	inline GPU_DEVICE GPU_CONSTEXPR random_device::result_type random_device::min()
	{
		return numeric_limits<result_type>::min();
	}

	inline GPU_DEVICE random_device::result_type random_device::operator()()
	{
		return hash<offset_t>()(threadIdx.x + blockIdx.x * blockDim.x);
	}
}
